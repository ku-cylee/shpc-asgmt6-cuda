#include "hip/hip_runtime.h"
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

#define GPU_THREADS   32
#define NUM_CYCLES    16

static int mpi_rank, mpi_world_size;

static __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                                     int K) {
  int local_row = threadIdx.y;
  int local_col = threadIdx.x;

  int global_row = GPU_THREADS * blockIdx.y + local_row;
  int global_col = GPU_THREADS * blockIdx.x + local_col;

  __shared__ float A_tile[GPU_THREADS][GPU_THREADS];
  __shared__ float B_tile[GPU_THREADS][GPU_THREADS];

  float sum = 0.0f;
  for (int k = 0; k < K; k += GPU_THREADS) {
    A_tile[local_row][local_col] = A[global_row * K + local_col + k];
    B_tile[local_row][local_col] = B[(local_row + k) * N + global_col];

    __syncthreads();

    for (int t = 0; t < GPU_THREADS; t++) {
      sum += A_tile[local_row][t] * B_tile[t][local_col];
    }

    __syncthreads();
  }

  C[global_row * N + global_col] = sum;
}

#define NGPU 4

static int Mbegin[NUM_CYCLES][NGPU], Mend[NUM_CYCLES][NGPU];
static int ngpu;
static hipStream_t streams[NGPU];
static float *A_gpu[NGPU], *B_gpu[NGPU], *C_gpu[NGPU];

void matmul(float *A, float *B, float *C, int M, int N, int K) {

  int M_per_cycle = M / NUM_CYCLES;
  int M_per_cycle_node = M_per_cycle / mpi_world_size;

  MPI_Request scatter_req, gather_req;
  
  MPI_Bcast(B, K * N, MPI_FLOAT, 0, MPI_COMM_WORLD);
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(B_gpu[i], B, K * N * sizeof(float),
                              hipMemcpyHostToDevice, streams[i]));
  }

  MPI_Iscatter(
    A, M_per_cycle_node * K, MPI_FLOAT,
    A, M_per_cycle_node * K, MPI_FLOAT,
    0, MPI_COMM_WORLD, &scatter_req);

  for (int cycle = 0; cycle < NUM_CYCLES; cycle++) {
    MPI_Wait(&scatter_req, MPI_STATUS_IGNORE);

    // Async memcpy H->D on each GPU
    for (int i = 0; i < ngpu; i++) {
      CHECK_CUDA(hipSetDevice(i));
      CHECK_CUDA(hipMemcpyAsync(A_gpu[i], &A[Mbegin[cycle][i] * K],
                                (Mend[cycle][i] - Mbegin[cycle][i]) * K * sizeof(float),
                                hipMemcpyHostToDevice, streams[i]));
    }

    if (cycle + 1 != NUM_CYCLES) {
      MPI_Iscatter(
        A + (cycle + 1) * M_per_cycle * K, M_per_cycle_node * K, MPI_FLOAT,
        A + (cycle + 1) * M_per_cycle * K, M_per_cycle_node * K, MPI_FLOAT,
        0, MPI_COMM_WORLD, &scatter_req);
    }

    // Run kernels asynchronously on each GPU
    for (int i = 0; i < ngpu; i++) {
      CHECK_CUDA(hipSetDevice(i));
      dim3 blockDim(GPU_THREADS, GPU_THREADS);
      dim3 gridDim((N + GPU_THREADS - 1) / GPU_THREADS, (Mend[cycle][i] - Mbegin[cycle][i] + GPU_THREADS - 1) / GPU_THREADS);
      matmul_kernel<<<gridDim, blockDim, 0, streams[i]>>>(
          A_gpu[i], B_gpu[i], C_gpu[i], Mend[cycle][i] - Mbegin[cycle][i], N, K);
      CHECK_CUDA(hipGetLastError());
    }

    if (cycle != 0) {
      MPI_Wait(&gather_req, MPI_STATUS_IGNORE);
    }

    // Async memcpy D->H on each GPU
    for (int i = 0; i < ngpu; i++) {
      CHECK_CUDA(hipSetDevice(i));
      CHECK_CUDA(hipMemcpyAsync(&C[Mbegin[cycle][i] * N], C_gpu[i],
                                (Mend[cycle][i] - Mbegin[cycle][i]) * N * sizeof(float),
                                hipMemcpyDeviceToHost, streams[i]));
    }

    // Wait for all async jobs to finish
    for (int i = 0; i < ngpu; i++) {
      hipSetDevice(i);
      hipStreamSynchronize(streams[i]);
    }

    MPI_Igather(
      C + cycle * M_per_cycle * N, M_per_cycle_node * N, MPI_FLOAT,
      C + cycle * M_per_cycle * N, M_per_cycle_node * N, MPI_FLOAT,
      0, MPI_COMM_WORLD, &gather_req);
  }

  MPI_Wait(&gather_req, MPI_STATUS_IGNORE);
  // MPI_Request_free(&scatter_req);
  // MPI_Request_free(&gather_req);
}


void matmul_initialize(int M, int N, int K) {
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  CHECK_CUDA(hipGetDeviceCount(&ngpu));

  printf("[rank %d] Number of devices: %d\n", mpi_rank, ngpu);
  hipDeviceProp_t props[NGPU];
  for (int i = 0; i < ngpu; ++i) {
    CHECK_CUDA(hipGetDeviceProperties(&props[i], i));
    printf("[rank %d] device %d: %s\n", mpi_rank, i, props[i].name);
  }

  int M_per_cycle = M / NUM_CYCLES;
  int M_per_node = M_per_cycle / mpi_world_size;
  int M_per_gpu = M_per_node / ngpu;

  for (int cycle = 0; cycle < NUM_CYCLES; cycle++) {
    for (int i = 0; i < ngpu; i++) {
      Mbegin[cycle][i] = cycle * M_per_cycle + M_per_gpu * i;
      Mend[cycle][i] = Mbegin[cycle][i] + M_per_gpu;
      // if (i == ngpu - 1) Mend[cycle][i] = M_per_node;
    }
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(
        hipMalloc(&A_gpu[i], M_per_gpu * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&B_gpu[i], K * N * sizeof(float)));
    CHECK_CUDA(
        hipMalloc(&C_gpu[i], M_per_gpu * N * sizeof(float)));
  }
}


void matmul_finalize() {
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipFree(A_gpu[i]));
    CHECK_CUDA(hipFree(B_gpu[i]));
    CHECK_CUDA(hipFree(C_gpu[i]));
    CHECK_CUDA(hipStreamDestroy(streams[i]));
  }
}
