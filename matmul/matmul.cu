#include "hip/hip_runtime.h"
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

#define TILE_SIZE     32
#define ITERATION     16

static int mpi_rank, mpi_world_size;

static __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                                     int K) {
  int local_row = threadIdx.y;
  int local_col = threadIdx.x;

  int global_row = TILE_SIZE * blockIdx.y + local_row;
  int global_col = TILE_SIZE * blockIdx.x + local_col;

  __shared__ float A_tile[TILE_SIZE][TILE_SIZE];
  __shared__ float B_tile[TILE_SIZE][TILE_SIZE];

  float sum = 0.0f;
  for (int k = 0; k < K; k += TILE_SIZE) {
    A_tile[local_row][local_col] = A[global_row * K + local_col + k];
    B_tile[local_row][local_col] = B[(local_row + k) * N + global_col];

    __syncthreads();

    for (int t = 0; t < TILE_SIZE; t++) {
      sum += A_tile[local_row][t] * B_tile[t][local_col];
    }

    __syncthreads();
  }

  C[global_row * N + global_col] = sum;
}

#define NGPU 4

static int Mbegin[ITERATION][NGPU], Mend[ITERATION][NGPU];
static int ngpu;
static hipStream_t streams[NGPU];
static float *A_gpu[NGPU], *B_gpu[NGPU], *C_gpu[NGPU];

void matmul(float *A, float *B, float *C, int M, int N, int K) {

  int M_per_iter = M / ITERATION;
  int M_per_iter_node = M_per_iter / mpi_world_size;

  MPI_Bcast(B, K * N, MPI_FLOAT, 0, MPI_COMM_WORLD);
  int start = 0, sc = 0, h2d = 0, mm = 0, d2h = 0, gt = 0;

  #pragma omp parallel num_threads(ITERATION)
  {
    int iter;

    #pragma omp critical
    {
      iter = start;
      start = iter + 1;
    }

    while (iter != 0) {
      int flag;
      #pragma omp atomic read
      flag = h2d;
      if (flag >= iter) break;
    }

    MPI_Scatter(
      A + iter * M_per_iter * N, M_per_iter_node * K, MPI_FLOAT,
      A + iter * M_per_iter * N, M_per_iter_node * K, MPI_FLOAT,
      0, MPI_COMM_WORLD);

    #pragma omp atomic write
    sc = iter + 1;
    while (iter != 0) {
      int flag;
      #pragma omp atomic read
      flag = mm;
      if (flag >= iter) break;
    }

    // Async memcpy H->D on each GPU
    for (int i = 0; i < ngpu; i++) {
      CHECK_CUDA(hipSetDevice(i));
      CHECK_CUDA(hipMemcpyAsync(A_gpu[i], &A[Mbegin[iter][i] * K],
                                (Mend[iter][i] - Mbegin[iter][i]) * K * sizeof(float),
                                hipMemcpyHostToDevice, streams[i]));
      if (iter == 0) {
        CHECK_CUDA(hipMemcpyAsync(B_gpu[i], B, K * N * sizeof(float),
                                  hipMemcpyHostToDevice, streams[i]));
      }
    }

    // Wait for all async jobs to finish
    for (int i = 0; i < ngpu; i++) {
      hipSetDevice(i);
      hipStreamSynchronize(streams[i]);
    }

    #pragma omp atomic write
    h2d = iter + 1;
    while (iter != 0) {
      int flag;
      #pragma omp atomic read
      flag = d2h;
      if (flag >= iter) break;
    }

    // Run kernels asynchronously on each GPU
    for (int i = 0; i < ngpu; i++) {
      CHECK_CUDA(hipSetDevice(i));
      dim3 blockDim(TILE_SIZE, TILE_SIZE);
      dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (Mend[iter][i] - Mbegin[iter][i] + TILE_SIZE - 1) / TILE_SIZE);
      matmul_kernel<<<gridDim, blockDim, 0, streams[i]>>>(
          A_gpu[i], B_gpu[i], C_gpu[i], Mend[iter][i] - Mbegin[iter][i], N, K);
      CHECK_CUDA(hipGetLastError());
    }

    // Wait for all async jobs to finish
    for (int i = 0; i < ngpu; i++) {
      hipSetDevice(i);
      hipStreamSynchronize(streams[i]);
    }

    #pragma omp atomic write
    mm = iter + 1;
    while (iter != 0) {
      int flag;
      #pragma omp atomic read
      flag = gt;
      if (flag >= iter) break;
    }

    // Async memcpy D->H on each GPU
    for (int i = 0; i < ngpu; i++) {
      CHECK_CUDA(hipSetDevice(i));
      CHECK_CUDA(hipMemcpyAsync(&C[Mbegin[iter][i] * N], C_gpu[i],
                                (Mend[iter][i] - Mbegin[iter][i]) * N * sizeof(float),
                                hipMemcpyDeviceToHost, streams[i]));
    }

    // Wait for all async jobs to finish
    for (int i = 0; i < ngpu; i++) {
      hipSetDevice(i);
      hipStreamSynchronize(streams[i]);
    }

    #pragma omp atomic write
    d2h = iter + 1;

    MPI_Gather(
      C + iter * M_per_iter * N, M_per_iter_node * N, MPI_FLOAT,
      C + iter * M_per_iter * N, M_per_iter_node * N, MPI_FLOAT,
      0, MPI_COMM_WORLD);

    #pragma omp atomic write
    gt = iter + 1;
  }
}


void matmul_initialize(int M, int N, int K) {
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  CHECK_CUDA(hipGetDeviceCount(&ngpu));

  printf("[rank %d] Number of devices: %d\n", mpi_rank, ngpu);
  hipDeviceProp_t props[NGPU];
  for (int i = 0; i < ngpu; ++i) {
    CHECK_CUDA(hipGetDeviceProperties(&props[i], i));
    printf("[rank %d] device %d: %s\n", mpi_rank, i, props[i].name);
  }

  int M_per_iter = M / ITERATION;
  int M_per_node = M_per_iter / mpi_world_size;
  int M_per_gpu = M_per_node / ngpu;

  for (int iter = 0; iter < ITERATION; iter++) {
    for (int i = 0; i < ngpu; i++) {
      Mbegin[iter][i] = iter * M_per_iter + i * M_per_gpu;
      Mend[iter][i] = Mbegin[iter][i] + M_per_gpu;
      // if (i == ngpu - 1) Mend[iter][i] = M_per_node;
    }
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(
        hipMalloc(&A_gpu[i], M_per_gpu * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&B_gpu[i], K * N * sizeof(float)));
    CHECK_CUDA(
        hipMalloc(&C_gpu[i], M_per_gpu * N * sizeof(float)));
  }
}


void matmul_finalize() {
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipFree(A_gpu[i]));
    CHECK_CUDA(hipFree(B_gpu[i]));
    CHECK_CUDA(hipFree(C_gpu[i]));
    CHECK_CUDA(hipStreamDestroy(streams[i]));
  }
}
