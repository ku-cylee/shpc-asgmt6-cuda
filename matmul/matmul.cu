#include "hip/hip_runtime.h"
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

#define TILE_SIZE     32

static int mpi_rank, mpi_world_size;

static __global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                                     int K) {
  int local_row = threadIdx.y;
  int local_col = threadIdx.x;

  int global_row = TILE_SIZE * blockIdx.y + local_row;
  int global_col = TILE_SIZE * blockIdx.x + local_col;

  __shared__ float A_tile[TILE_SIZE][TILE_SIZE];
  __shared__ float B_tile[TILE_SIZE][TILE_SIZE];

  float sum = 0.0f;
  for (int k = 0; k < K; k += TILE_SIZE) {
    A_tile[local_row][local_col] = A[global_row * K + local_col + k];
    B_tile[local_row][local_col] = B[(local_row + k) * N + global_col];

    __syncthreads();

    for (int t = 0; t < TILE_SIZE; t++) {
      sum += A_tile[local_row][t] * B_tile[t][local_col];
    }

    __syncthreads();
  }

  C[global_row * N + global_col] = sum;
}

#define NGPU 4

static int Mbegin[NGPU], Mend[NGPU];
static int ngpu;
static hipStream_t streams[NGPU];
static float *A_gpu[NGPU], *B_gpu[NGPU], *C_gpu[NGPU];


void matmul(float *A, float *B, float *C, int M, int N, int K) {

  int M_per_node = M / mpi_world_size;

  MPI_Scatter(
    A, M_per_node * K, MPI_FLOAT,
    A, M_per_node * K, MPI_FLOAT,
    0, MPI_COMM_WORLD);
  MPI_Bcast(B, K * N, MPI_FLOAT, 0, MPI_COMM_WORLD);

  // Async memcpy H->D on each GPU
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(A_gpu[i], &A[Mbegin[i] * K],
                               (Mend[i] - Mbegin[i]) * K * sizeof(float),
                               hipMemcpyHostToDevice, streams[i]));
    CHECK_CUDA(hipMemcpyAsync(B_gpu[i], B, K * N * sizeof(float),
                               hipMemcpyHostToDevice, streams[i]));
  }

  // Run kernels asynchronously on each GPU
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (Mend[i] - Mbegin[i] + TILE_SIZE - 1) / TILE_SIZE);
    matmul_kernel<<<gridDim, blockDim, 0, streams[i]>>>(
        A_gpu[i], B_gpu[i], C_gpu[i], Mend[i] - Mbegin[i], N, K);
    CHECK_CUDA(hipGetLastError());
  }

  // Async memcpy D->H on each GPU
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipMemcpyAsync(&C[Mbegin[i] * N], C_gpu[i],
                               (Mend[i] - Mbegin[i]) * N * sizeof(float),
                               hipMemcpyDeviceToHost, streams[i]));
  }

  // Wait for all async jobs to finish
  for (int i = 0; i < ngpu; i++) {
    hipSetDevice(i);
    hipStreamSynchronize(streams[i]);
  }

  MPI_Gather(
    C, M_per_node * N, MPI_FLOAT,
    C, M_per_node * N, MPI_FLOAT,
    0, MPI_COMM_WORLD);
}


void matmul_initialize(int M, int N, int K) {
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  CHECK_CUDA(hipGetDeviceCount(&ngpu));

  printf("[rank %d] Number of devices: %d\n", mpi_rank, ngpu);
  hipDeviceProp_t props[NGPU];
  for (int i = 0; i < ngpu; ++i) {
    CHECK_CUDA(hipGetDeviceProperties(&props[i], i));
    printf("[rank %d] device %d: %s\n", mpi_rank, i, props[i].name);
  }

  int M_per_node = M / mpi_world_size;
  int M_per_gpu = M_per_node / ngpu;

  for (int i = 0; i < ngpu; i++) {
    Mbegin[i] = M_per_gpu * i;
    Mend[i] = Mbegin[i] + M_per_gpu;
    if (i == ngpu - 1) Mend[i] = M_per_node;
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipStreamCreate(&streams[i]));
  }

  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(
        hipMalloc(&A_gpu[i], (Mend[i] - Mbegin[i]) * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&B_gpu[i], K * N * sizeof(float)));
    CHECK_CUDA(
        hipMalloc(&C_gpu[i], (Mend[i] - Mbegin[i]) * N * sizeof(float)));
  }
}


void matmul_finalize() {
  for (int i = 0; i < ngpu; i++) {
    CHECK_CUDA(hipSetDevice(i));
    CHECK_CUDA(hipFree(A_gpu[i]));
    CHECK_CUDA(hipFree(B_gpu[i]));
    CHECK_CUDA(hipFree(C_gpu[i]));
    CHECK_CUDA(hipStreamDestroy(streams[i]));
  }
}
